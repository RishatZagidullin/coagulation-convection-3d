#include "hip/hip_runtime.h"
#include "cuda_gl.cuh"
#include <iostream>
#include <algorithm>

namespace gl
{
    //all that needs to change is the function bellow and ctor args
    //to fully variate the interop behaviour I think.
    //The rest can stay the same.
    template<typename T>
    __global__ void map_to_gpu(T * output, T * input, int size, int ind)
    {
        int numThreads = blockDim.x * gridDim.x;
        int global_id = threadIdx.x + blockIdx.x * blockDim.x;

        for (int id = global_id; id < size; id+=numThreads)
        {
            output[6*(id)+3+ind] = input[id];
        }
        return;
    }

    template<typename T>
    void interop<T>::update_gpu_data(T * data, int ind)
    {
        hipMemcpy(data_device, data, sizeof(T)*size,
                   hipMemcpyHostToDevice);

        map_to_gpu<<<grid, block>>>(gl_data, data_device, size, ind);
        hipDeviceSynchronize();
    }

    template<typename T>
    interop<T>::interop(unsigned int * BO, int size)
    {
        this->size = size;
        block = 256;
        grid = (size + block.x-1)/block.x;
        hipMalloc(&data_device, sizeof(T)*size);
        hipGraphicsGLRegisterBuffer(&gl_buffer, *BO,
                                 cudaGraphicsMapFlagsNone);
        hipGraphicsMapResources(1, &gl_buffer, 0);
        hipGraphicsResourceGetMappedPointer((void**)&gl_data,
                   nullptr, gl_buffer);
    }

    template<typename T>
    interop<T>::~interop()
    {
        hipGraphicsUnmapResources(1, &gl_buffer, 0);
        hipFree(data_device);
    }

    template class interop<float>;
}
