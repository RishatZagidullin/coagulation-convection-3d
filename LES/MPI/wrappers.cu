#include "hip/hip_runtime.h"
#include "wrappers.h"

namespace wrappers
{
    __global__ void calc_smoluch(const double *L1, const double *L2, 
                                 double *n_k, int N, double dt)
    {
        const int numThreads = blockDim.x * gridDim.x;
        const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
        for (int i = threadID; i < N; i += numThreads)
        {
            //mod is cause you don't call bubble from smol_conv_discrete
            double temp = n_k[i];
            n_k[i] = ( L1[(N+i-1)%N] * 0.5 - temp * L2[i] ) * dt + temp;
        }
    }
}

