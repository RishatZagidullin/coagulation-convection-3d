#include "glad/glad.h"
#include "utils/utils.cuh"
#include "solvers/mpi_solver.h"
#include "geometry/projection_reg.h"
#include "gl/viewer.h"
#include "gl/grid_viewer.h"
#include "gl/cuda_gl.cuh"
#include "gl/animation_reg.h"
#include <time.h>
#include <sys/time.h>
#include "geometry/parse_foam.h"
#include "wrappers.h"
#include <mpi.h>

void smol_iteration_fft_c(solvers::mpi_solver<double> & solver, int S,
          double *& n_k, double *& L1, double *& L2,
          fftw_complex *& ub, fftw_complex *& vb, double dt,
          TCross_Parallel_v1 & crossed_kernel,
          fftw_plan *& plan_v, fftw_plan *& plan_u, fftw_plan *& plan_inv)
{
    for (int i = 0; i < solver.N_; i++)
        for (int j = 0; j < solver.M_; j++)
            for (int k = 0; k < solver.K_; k++)
    {
        //double st = MPI_Wtime();
        int x = i*solver.M_*solver.K_+j*solver.K_+k;
        for (int m = 0; m < S; m++)
        {
            int ind = x+solver.M_*solver.N_*solver.K_*m;
            n_k[m] = solver.data[ind];
        }
        L2 = crossed_kernel.matvec(n_k);
        L1 = crossed_kernel.smol_conv_discrete(n_k, ub, vb, 
                                             plan_v, plan_u, plan_inv);

        for (int m = 0; m < S; m++)
        {
            int ind = x+solver.M_*solver.N_*solver.K_*m;
            solver.data_new[ind] = ( L1[m] * 0.5 -
                                 n_k[m] * L2[m]) * dt + n_k[m];
            if (solver.data_new[ind] < 0.0) solver.data_new[ind] = 0.0;
        }
        delete [] L2;
        delete [] L1;
    }
}

void smol_iteration_fft_g(solvers::mpi_solver<double> & solver, int S,
          double *& n_k, double *& L1, double *& L2,
          hipblasHandle_t & handle, hipfftHandle & plan, double dt,
          TCross_Parallel_v1 & crossed_kernel)
{
    dim3 bl(128);
    dim3 gr((S+bl.x-1)/bl.x);
    for (int i = 0; i < solver.N_; i++)
        for (int j = 0; j < solver.M_; j++)
            for (int k = 0; k < solver.K_; k++)
    {
        
        int x = i*solver.M_*solver.K_+j*solver.K_+k;
        for (int m = 0; m < S; m++)
        {
            int ind = x+solver.M_*solver.N_*solver.K_*m;
            n_k[m] = solver.data[ind];
        }

        crossed_kernel.matvec(n_k, handle, L2);
        crossed_kernel.smol_conv_discrete(n_k, plan, handle, L1);

        checkCudaErrors(hipDeviceSynchronize());
        wrappers::calc_smoluch<<<gr, bl>>>(L1, L2, n_k, S, dt);
        checkCudaErrors(hipDeviceSynchronize());

        for (int m = 0; m < S; m++)
        {
            int ind = x+solver.M_*solver.N_*solver.K_*m;
            solver.data_new[ind] = n_k[m];
            if (solver.data_new[ind] < 0.0) solver.data_new[ind] = 0.0;
        }
    }
}

//from no on: x (N) - progragation, z (M) - up/down

//hardcodes:
//animation_reg.h update_proj_colors N/6 and 0/2
//projection_reg.h init_data -3.0
//advection_reg.cpp find_foam +2.0 +2.0 +3.0
//advection_reg.cpp find_foam foam_dx,foam_dy...
//advection_reg.cpp upd_downward_velo N/6 and K/2
//diffusion_reg.cpp st_point.z
//diffusion_reg.cpp source location (0.,0.)
//think that's it but not sure

int main(int argc, char ** argv)
{
    MPI_Init(&argc, &argv);
    int size;
    int rank;
    MPI_Comm comm = MPI_COMM_WORLD;
    MPI_Comm_size(comm, &size);
    MPI_Comm_rank(comm, &rank);

    double start = MPI_Wtime();
    int n_vort = 288000;
    parse_foam<100> foam("./grid", n_vort);

    grid_params proj_grid = grid_params(10, 15, 30,
                                 -2.0, -3.0, -2.0, 0.4);
    short SCR_W = 1600;
    short SCR_H = 1200;
    int N = proj_grid.h;
    int M = proj_grid.w;
    int K = proj_grid.d;
    double dd = proj_grid.dd;
    int S = 8;
    double dt = 0.001;
    //assert that size == X*Y*Z
    int X = 1;
    int Y = 1;
    int Z = 1;
    if (rank == 0)
    {
        std::cout << "SIZE: " << size << "\n";
        std::cout << "X: " << X << "\n";
        std::cout << "Y: " << Y << "\n";
        std::cout << "Z: " << Z << "\n";
    }

    //***************GPU STUFF START*******************
    double tolerance = 1e-4;    			
    TCross_Parallel_v1 crossed_kernel = wrappers::default_crossed_kernel(tolerance, S, 1);		

    double R_value = crossed_kernel.get_rank();
    double V_value = crossed_kernel.get_columns_number();
    std::cout << "R value: " << R_value << "\n";
    std::cout << "V value: " << V_value << "\n";

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int Rnum = crossed_kernel.get_rows_number();
    int n[] = { Rnum }; 
    int inembed[] = { 0 };
    int onembed[] = { 0 }; 
    hipfftHandle plan;
    hipfftPlanMany(&plan, 1, n, inembed, 1, Rnum,
                  onembed, 1, Rnum, HIPFFT_Z2Z, R_value);

    double *n_k;
    gpuErrchk(hipMallocManaged(&n_k, S*sizeof(double)));
    //n_k = new double [S];
    double *L1_res_g, *L2_res_g;
    gpuErrchk(hipMallocManaged(&L1_res_g, S*sizeof(double)));
    gpuErrchk(hipMallocManaged(&L2_res_g, S*sizeof(double)));
    //***************GPU STUFF END*********************

        //****************LOW RANK STUFF START*************
    fftw_complex *ub = (fftw_complex *) fftw_malloc(R_value * V_value * sizeof(fftw_complex));
    fftw_complex *vb = (fftw_complex *) fftw_malloc(R_value * V_value * sizeof(fftw_complex));
    fftw_plan * plan_v = (fftw_plan *) fftw_malloc(R_value * sizeof(fftw_plan));
    fftw_plan * plan_u = (fftw_plan *) fftw_malloc(R_value * sizeof(fftw_plan));
    fftw_plan * plan_inv = (fftw_plan *) fftw_malloc(R_value * sizeof(fftw_plan));
    for (int i = 0; i < R_value; i++)
    {
        plan_v[i] = fftw_plan_dft_1d(S, vb+i*S, vb+i*S, FFTW_FORWARD, FFTW_ESTIMATE);
        plan_u[i] = fftw_plan_dft_1d(S, ub+i*S, ub+i*S, FFTW_FORWARD, FFTW_ESTIMATE);
        plan_inv[i] = fftw_plan_dft_1d(S, ub+i*S, ub+i*S, FFTW_BACKWARD, FFTW_ESTIMATE);	
    }
    double *L1_res_c, *L2_res_c;
    //****************LOW RANK STUFF END***************


    MPI_Barrier(MPI_COMM_WORLD);
    double D = 0.01;
    solvers::mpi_solver<double> solver(foam, rank, dd, dt, D,
                                      N/X, M/Z, K/Y, S, X, Y, Z);

    double * data, * data_recv;
    if (rank == 0)
    {
        data = new double [N*M*K*S];
        data_recv = new double [N/X*M/Z*K/Y*S];
    }
    else
    {
        data = new double [N/X*M/Z*K/Y*S];
    }

    AnimationData_reg * colors;
    constexpr int num_objs = 1;
    gl::viewer<num_objs, gl::grid_viewer> * v;
    gl::interop<float> * scalar_interop;
    projection_reg * proj;
    if (rank == 0)
    {
        proj = new projection_reg(proj_grid);
        //true is to hide window
        v = new gl::viewer<num_objs, gl::grid_viewer>(SCR_W, SCR_H, true);
        colors = new AnimationData_reg(N,M,K);
        v->buffer<GL_ARRAY_BUFFER>(proj->max_v*6,proj->vertices,0);
        v->buffer<GL_ELEMENT_ARRAY_BUFFER>(proj->max_i*6, 
                                           proj->indices, 1);
        scalar_interop = new gl::interop<float>(&v->buffer_objects[0],
                                                proj->max_v);
    }

    //int time = 0;
    int TIME_MAX = 1000;
    MPI_Barrier(MPI_COMM_WORLD);
    if (rank == 0)
    {
        std::cout << "Preprocessing time: " << MPI_Wtime()-start;
        std::cout << "\n";
    }
    start = MPI_Wtime();
    //int ccount = 0;
    for (int time = 0; time < TIME_MAX; time++)
    {
        if (time % 5 == 1)
        {

            if (rank == 0)
            {
                for (int s = 0; s < S; s++)
                    for (int i = 0; i < N/X; i++)
                        for (int j = 0; j < M/Z; j++)
                            for (int k = 0; k < K/Y; k++)
                {
                    int ind = k+1 + (j+1)*solver.K_ +
                              (i+1)*solver.K_*solver.M_ +
                              s*solver.K_*solver.M_*solver.N_;

                    data[k+j*K+i*K*M+s*K*M*N] = solver.data[ind];
                }
                int n = 0;
                for (int x = 0; x < X; x++)
                    for (int z = 0; z < Z; z++)
                        for (int y = 0; y < Y; y++)
                {
                    if (n > 0)
                    {
                        MPI_Recv(data_recv, N/X*M/Z*K/Y*S, MPI_DOUBLE,
                             n, n, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
                        for (int s = 0; s < S; s++)
                            for (int i = 0; i < N/X; i++)
                                for (int j = 0; j < M/Z; j++)
                                    for (int k = 0; k < K/Y; k++)
                        {
                            int ind = k + y*K/Y + (j + z*M/Z) * K + (i + x*N/X) * M*K + s * N*M*K;
                            data[ind] = data_recv[k+j*K/Y+i*K/Y*M/Z+s*K/Y*M/Z*N/X];
                        }
                    }
                    n++;
                }
                colors->update_proj_colors(data+N*M*K*1);
                scalar_interop->update_gpu_data(colors->colors_proj, 0);
                colors->update_proj_colors(data+N*M*K*3);
                scalar_interop->update_gpu_data(colors->colors_proj, 1);
                colors->update_proj_colors(data+N*M*K*7);
                scalar_interop->update_gpu_data(colors->colors_proj, 2);

                int obj_lens [num_objs] = {proj->max_i*6};
                //do not pass the time argument 
                //if you do not want any images to be saved
                v->view(obj_lens, time);
            }
            else
            {
                for (int s = 0; s < S; s++)
                    for (int i = 0; i < N/X; i++)
                        for (int j = 0; j < M/Z; j++)
                            for (int k = 0; k < K/Y; k++)
                {
                    int ind = k+1 + (j+1)*solver.K_ +
                                    (i+1)*solver.K_*solver.M_ +
                                    s*solver.K_*solver.M_*solver.N_;

                    data[k+j*K/Y+i*K/Y*M/Z+s*K/Y*M/Z*N/X] = 
                                    solver.data[ind];
                }
                MPI_Send(data, N/X*M/Z*K/Y*S, MPI_DOUBLE, 0, rank,
                         MPI_COMM_WORLD);
            }

            MPI_Barrier(MPI_COMM_WORLD);
        }

        smol_iteration_fft_g(solver, S, n_k, L1_res_g, 
                           L2_res_g, handle, plan, dt, 
                           crossed_kernel);

        //smol_iteration_fft_c(solver, S, n_k, L1_res_c, 
        //                   L2_res_c, ub, vb, dt, crossed_kernel,
        //                   plan_v, plan_u, plan_inv);

        hipDeviceSynchronize();
        solver.iteration(time);
        if (size>1)
            solver.exchange();

        //printProgress((double)time/TIME_MAX);
    }
    MPI_Barrier(MPI_COMM_WORLD);
    if (rank == 0)
    {
        std::cout << "Processing time: " << MPI_Wtime()-start << endl;
    }
    hipFree(n_k);
    hipFree(L1_res_g);
    hipFree(L2_res_g);
    hipblasDestroy(handle);

    for (int i = 0; i < R_value; i++)
    {
        fftw_destroy_plan(plan_v[i]);
        fftw_destroy_plan(plan_u[i]);
        fftw_destroy_plan(plan_inv[i]);	
    }
    fftw_free(vb);
    fftw_free(ub);
    fftw_free(plan_u);
    fftw_free(plan_v);
    fftw_free(plan_inv);

    MPI_Finalize();
    return 0;
}
